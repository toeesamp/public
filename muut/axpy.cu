
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

#define MAXN 30000000
#define A (-1.0)
#define B (1.0)

//säieryhmän koko
#define LOCAL_SIZE 1024

//säieryhmien määrä
//#define WG_COUNT (MAXN/LOCAL_SIZE+1)


//ydin, jolle annetaan kaksi n kokoista vektoria x ja y, sekä
//liukuluku m, joille suoritetaan axpy-operaatio
__global__ void axpy(double *x, double *y, double m, int n) {

	const int global_id = blockIdx.x * blockDim.x + threadIdx.x;


	if(global_id < n)
		y[global_id] = m * x[global_id] + y[global_id];

}


//tarkistetaan onko doublet samoja
bool double_equals( double a, double b, double epsilon = 0.0001) {

	return std::abs(a-b) < epsilon;
}

int main() {

	hipError_t err;

	// Alustetaan satunnaislukugeneraattori kellonajalla
	srand(time(NULL));

	// Satunnainen kokonaisluku väliltä [1,MAXN]
	int n = rand() % MAXN + 1;

	// Varataan tilaa vektoreille ja luodaan satunnainen liukuluku
	double *x = new double[n];
	double *y = new double[n];
	double m = (B-A)*((double)rand()/RAND_MAX)+A;

	//ja vektorit testaamista varten
	double *x_test = new double[n];
	double *y_test = new double[n];


	// Generoidaan kaksi satunnaista vektoria
	for(int i = 0; i < n; i++) {
		x[i] = (B-A)*((double)rand()/RAND_MAX)+A;
		y[i] = (B-A)*((double)rand()/RAND_MAX)+A;
	}

	//kopioidaan vektorit talteen testaamista varten
	for(int i = 0; i < n; i++) {
		x_test[i] = x[i];
		y_test[i] = y[i];
	}

	//Varataan muisti Cuda-laitteelta

	double *deviceBuffer1, *deviceBuffer2;

	//Varataan deviceBuffer1
	err = hipMalloc((void **)&deviceBuffer1, n*sizeof(double));

	if(err != hipSuccess) {
		std::cout << "Muistin varaaminen epäonnistui." << std::endl;

		// cudaError on union-tietotyyppi. Virhekoodia vastaava virheviestijono
		// saadaan selville cudaGetErrorString-aliohjelman avulla.
		std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
			std::endl;
		delete [] x;
		delete [] y;
		delete [] x_test;
		delete [] y_test;
		return 1;
	}

	//Varataan deviceBuffer2
	err = hipMalloc((void **)&deviceBuffer2, n*sizeof(double));

	if(err != hipSuccess) {
		std::cout << "Muistin varaaminen epäonnistui." << std::endl;

		// cudaError on union-tietotyyppi. Virhekoodia vastaava virheviestijono
		// saadaan selville cudaGetErrorString-aliohjelman avulla.
		std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
			std::endl;
			delete [] x;
			delete [] y;
			delete [] x_test;
			delete [] y_test;
		return 1;
	}

	//Siirretään data cuda-laitteelle

  	//Siirretään x -> deviceBuffer1
	err = hipMemcpy(
		deviceBuffer1, x, n*sizeof(double), hipMemcpyHostToDevice);

	if(err != hipSuccess) {
		std::cout << "Isäntälaite -> GPU -siirtokäskyn asettaminen " \
			"komentojonoon epäonnistui." << std::endl;
		std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
			std::endl;

		hipFree(deviceBuffer1);
		hipFree(deviceBuffer2);
		delete [] x;
		delete [] y;
		delete [] x_test;
		delete [] y_test;
		return 1;
	}

	//Siirretään y -> deviceBuffer2
	err = hipMemcpy(
		deviceBuffer2, y, n*sizeof(double), hipMemcpyHostToDevice);

	if(err != hipSuccess) {
		std::cout << "Isäntälaite -> GPU -siirtokäskyn asettaminen " \
			"komentojonoon epäonnistui." << std::endl;
		std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
			std::endl;

		hipFree(deviceBuffer1);
		hipFree(deviceBuffer2);
		delete [] x;
		delete [] y;
		delete [] x_test;
		delete [] y_test;
		return 1;
	}

	//ytimen käynnistäminen

	// Säieryhmien määrä
	dim3 WGCount(n/LOCAL_SIZE+1, 1, 1);

	// Lokaalin indeksiavaruuden koko
	dim3 localDim(LOCAL_SIZE, 1, 1);

	axpy<<<WGCount, localDim>>>(deviceBuffer1, deviceBuffer2, m, n);

	//kysytään tuliko ytimen käynnistämisessä virheitä
	err = hipGetLastError();

	if(err != hipSuccess) {
		std::cout << "Ytimen käynnistyskäskyn asettaminen komentojonoon " \
			"epäonnistui." << std::endl;
		std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
			std::endl;

		hipFree(deviceBuffer1);
		hipFree(deviceBuffer2);
		delete [] x;
		delete [] y;
		delete [] x_test;
		delete [] y_test;
		return 1;
	}

	//
	// Siirretään data takaisin isäntälaiteen muistiin
	//

	//x ja deviceBuffer1
	hipMemcpy(x, deviceBuffer1, n*sizeof(double), hipMemcpyDeviceToHost);

	if(err != hipSuccess) {
		std::cout << "GPU -> Isäntälaite -siirtokäskyn asettaminen " \
			"komentojonoon epäonnistui." << std::endl;
		std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
			std::endl;

			hipFree(deviceBuffer1);
			hipFree(deviceBuffer2);
			delete [] x;
			delete [] y;
			delete [] x_test;
			delete [] y_test;
		return 1;
	}

	//y ja deviceBuffer2
	hipMemcpy(y, deviceBuffer2, n*sizeof(double), hipMemcpyDeviceToHost);

		if(err != hipSuccess) {
			std::cout << "GPU -> Isäntälaite -siirtokäskyn asettaminen " \
				"komentojonoon epäonnistui." << std::endl;
			std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
				std::endl;

			hipFree(deviceBuffer1);
			hipFree(deviceBuffer2);
			delete [] x;
			delete [] y;
			delete [] x_test;
			delete [] y_test;
			return 1;
		}

		//
		// Tarkistetaan tulos
		//


		//lasketaan axpy cpu:lla
		for(int i = 0; i < n; i++) {
			y_test[i] = m * x_test[i] + y_test[i];
		}

		//tarkistetaan ovatko kaikki y:n arvot samoja
		bool correct = true;
		for(int i = 0; i < n; i++) {
			if (double_equals(y_test[i], y[i]) != true) {
				correct = false;
				//tulostetaan eriävät arvot
				std::cout << y_test[i] << std::endl;
				std::cout << y[i] << std::endl;
				break;
			}
		}

		std::cout << "The result was " ;
		if (!correct) {
			std::cout << "incorrect." << std::endl;
		}
		else std::cout << "correct." << std::endl;


		//
		// Vapautetaan CUDA-laitteen puolelta varattu muisti
		//
		err = hipFree(deviceBuffer1);

		if(err != hipSuccess) {
			std::cout << "Muistin vapauttaminen epäonnistui epäonnistui." <<
				std::endl;
			std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
				std::endl;

			delete [] x;
			delete [] y;
			delete [] x_test;
			delete [] y_test;
			return 1;
		}

		err = hipFree(deviceBuffer2);

		if(err != hipSuccess) {
			std::cout << "Muistin vapauttaminen epäonnistui epäonnistui." <<
				std::endl;
			std::cerr << "CUDA-virhekoodi: " << hipGetErrorString(err) <<
				std::endl;

			delete [] x;
			delete [] y;
			delete [] x_test;
			delete [] y_test;
			return 1;
		}

		delete [] x;
		delete [] y;
		delete [] x_test;
		delete [] y_test;

		return 0;
}
